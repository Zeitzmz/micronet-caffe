#include "hip/hip_runtime.h"
/*
 * Quantize Layer
 *
 * Created on: Aug 19, 2019
 * Author: hujie (Momenta)
 */

#include <cfloat>

#include "caffe/proto/caffe.pb.h"
#include "caffe/layers/quantize_layer.hpp"
#include "caffe/util/gpu_util.cuh"

namespace caffe {

#define STATS_BINS 2048

__device__ unsigned int counter = 0;

template <typename Dtype>
static __global__ void QuantizeGetHist(const Dtype* data, int count,
    Dtype* workspace, int num_bins, Dtype* hist, Dtype* src_step) {
  // Get absolute max to calculate src_step
  __shared__ Dtype max_buffer[CAFFE_CUDA_NUM_THREADS];
  __shared__ bool is_lastblock_done; // flag refer to whether the last block
  max_buffer[threadIdx.x] = FLT_MIN;
  for (unsigned int gid = blockDim.x * blockIdx.x + threadIdx.x;
      gid < count; gid += blockDim.x * gridDim.x) {
    max_buffer[threadIdx.x] = max(max_buffer[threadIdx.x], abs(data[gid]));
  }
  __syncthreads();
    
  for (unsigned int i = blockDim.x / 2; i > 0; i >>= 1) {
    int gid = blockDim.x * blockIdx.x + threadIdx.x;
    if (threadIdx.x < i && gid < count) {
      max_buffer[threadIdx.x] = max(max_buffer[threadIdx.x], max_buffer[threadIdx.x + i]);
    }
    __syncthreads();
  }
  
  if (threadIdx.x == 0) {
    workspace[blockIdx.x] = max_buffer[0];
    __threadfence();
    unsigned int value = atomicInc(&counter, gridDim.x); // accumulate how many blocks have down
    is_lastblock_done = (value == (gridDim.x - 1));
  }
  __syncthreads();

  // Use last block to conduct max reduce across blocks.
  if (is_lastblock_done) {
    max_buffer[threadIdx.x] = FLT_MIN;
    for (unsigned int index = threadIdx.x; index < gridDim.x; index += blockDim.x) {
      max_buffer[threadIdx.x] = max(max_buffer[threadIdx.x], workspace[index]);
    }
    __syncthreads();
    
    for (unsigned int i = blockDim.x / 2; i > 0; i >>= 1) {
      if (threadIdx.x < i) {
        max_buffer[threadIdx.x] = max(max_buffer[threadIdx.x], max_buffer[threadIdx.x + i]);
      }
      __syncthreads();
    }
    if (threadIdx.x == 0) {
      *src_step = max_buffer[0] / STATS_BINS;  // write down src_step
    }
  }
  __syncthreads();

  // Generate data distribution
  Dtype step = *src_step; 
  __shared__ Dtype hist_buffer[STATS_BINS];
  for (unsigned int i = threadIdx.x; i < num_bins; i += blockDim.x) {
    hist_buffer[i] = 0;
  }
  
  // generate hist within block     
  for (unsigned int gid = blockDim.x * blockIdx.x + threadIdx.x;
      gid < count; gid += blockDim.x * gridDim.x) {
    Dtype abs_data = abs(data[gid]);
    if (abs_data > Dtype(0.0001)) {
      int bin_index = max(abs_data / step, Dtype(num_bins - 1));
      caffe_gpu_atomic_add(Dtype(1), hist_buffer + bin_index);
    }
  }

  // accumulate hist across blocks
  __threadfence(); 
  for (int i = threadIdx.x; i < num_bins; i += blockDim.x) {
    caffe_gpu_atomic_add(hist_buffer[i], hist+i);
  }
}

template <typename Dtype>
static __global__ void QuantizeMinimizeKLDivs(const Dtype* hist, int num_bins, 
    int num_quant_bins, int num_kl_divs, Dtype tolerance, Dtype* kl_divs, 
    const Dtype* src_step, Dtype* final_step) {
  __shared__ Dtype kl_buffer[CAFFE_CUDA_NUM_THREADS];
  __shared__ bool is_lastblock_done; // flag refer to whether the last block
  kl_buffer[threadIdx.x] = FLT_MAX;

  for (int i = threadIdx.x; i < num_quant_bins; i += blockDim.x) {
    int gid = blockIdx.x * blockDim.x + i;
    if (gid >= num_kl_divs * num_quant_bins) 
      break;
    int kl_index = gid / num_quant_bins;
    int bin_index = gid % num_quant_bins;
    
    Dtype num_merged_bins = Dtype(num_quant_bins + kl_index) / num_quant_bins;
    Dtype start_not_rounded = num_merged_bins * bin_index;
    Dtype end_not_rounded = start_not_rounded + num_merged_bins;

    int start = floor(start_not_rounded);
    int end = ceil(end_not_rounded);
    start = min(start, num_bins - 1);
    end = min(end, num_bins);
    if (bin_index == num_quant_bins - 1) {
      end_not_rounded = num_bins;
      end = num_bins;
    }

    float non_zero_len = 0; // Use decimal fraction to precise equipartition
    float sum = 0;
    for (int j = start; j < end; ++j) {
      Dtype fraction = 1;
      if (j == start) {
        fraction = start + 1 - start_not_rounded;
      } else if (j == end - 1) {
        fraction = end_not_rounded - (end - 1);
      }
      if (hist[j] != Dtype(0)) {
        sum += hist[j] * fraction;
        non_zero_len += fraction;
      }
    }
    
    for (int j = start; j < end; ++j) {
      Dtype fraction = 1;
      if (j == start) {
        fraction = start + 1 - start_not_rounded;
      } else if (j == end - 1) {
        fraction = end_not_rounded - (end - 1);
      }
      if (hist[j] != Dtype(0)) {
        kl_buffer[threadIdx.x] += hist[j] * fraction 
            * log(hist[j] / (sum / non_zero_len));
      }
    }
  }
  __syncthreads();

  for (int i = blockDim.x / 2; i > 0; i >>= 1) {
    int index = blockIdx.x * blockDim.x + i;
    if (threadIdx.x < i && index < num_kl_divs * num_quant_bins) {
      kl_buffer[threadIdx.x] += kl_buffer[threadIdx.x + i];
    }
    __syncthreads();
  }
  
  if (threadIdx.x == 0) {
    kl_divs[blockIdx.x] = kl_buffer[0];
    __threadfence();
    unsigned int value = atomicInc(&counter, gridDim.x);
    is_lastblock_done = (value == (gridDim.x - 1));
  }
  __syncthreads();

  // Use last block to conduct min reduce across kl_divs.
  if (is_lastblock_done) {
    kl_buffer[threadIdx.x] = FLT_MAX;
    for (unsigned int index = threadIdx.x; index < gridDim.x; index += blockDim.x) {
      kl_buffer[threadIdx.x] = min(kl_buffer[threadIdx.x], kl_divs[index]);
    }
    __syncthreads();
    
    for (unsigned int i = blockDim.x / 2; i > 0; i >>= 1) {
      if (threadIdx.x < i) {
        kl_buffer[threadIdx.x] = min(kl_buffer[threadIdx.x], kl_buffer[threadIdx.x + i]);
      }
      __syncthreads();
    }
    
    // find the max index whose KL <= min_kl * tolerance
    Dtype loose_kl = kl_buffer[0] * tolerance;
    __shared__ unsigned int max_index[CAFFE_CUDA_NUM_THREADS];
    max_index[threadIdx.x] = 0;
    for (unsigned int index = threadIdx.x; index < gridDim.x; index += blockDim.x) {
      if (kl_divs[index] <= loose_kl && index > max_index[threadIdx.x]) {
        max_index[threadIdx.x] = index;
      }
    }
    for (unsigned int i = blockDim.x / 2; i > 0; i >>= 1) {
      if (threadIdx.x < i) {
        max_index[threadIdx.x] = max(max_index[threadIdx.x], max_index[threadIdx.x + i]);
      }
      __syncthreads();
    }
    if (threadIdx.x == 0) {
      *final_step = (num_quant_bins + max_index[0] + 0.5) * src_step[0] / num_quant_bins;
    }
  }
}

template<typename Dtype>
static __global__ void Quantize(const Dtype* input, int count, 
    Dtype step, Dtype min_val, Dtype max_val, Dtype* output) {
  CUDA_KERNEL_LOOP(index, count) {
    output[index] = min(max(round(input[index] / step) * step, max_val), min_val);
  }
}

template <typename Dtype>
void QuantizeLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  int count = bottom[0]->count();
  if (this->phase_ == TRAIN && !frozen_) {
    workspace_.Reshape(vector<int>(1, CAFFE_GET_BLOCKS(count)));
    hist_.Reshape(vector<int>(1, STATS_BINS));

    QuantizeGetHist<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        bottom[0]->gpu_data(), 
        count,
        workspace_.mutable_gpu_data(), 
        STATS_BINS, 
        hist_.mutable_gpu_data(),
        this->blobs_[0]->mutable_gpu_data());
    CUDA_POST_KERNEL_CHECK;

    int num_quant_bins = positive_?(1 << precision_):(1 << (precision_ - 1));
    kl_divs_.Reshape(vector<int>(1, STATS_BINS - num_quant_bins));
    QuantizeMinimizeKLDivs<Dtype><<<STATS_BINS - num_quant_bins, CAFFE_CUDA_NUM_THREADS>>>(
        hist_.gpu_data(), 
        STATS_BINS,
        num_quant_bins, 
        STATS_BINS - num_quant_bins, 
        tolerance_,
        kl_divs_.mutable_gpu_data(),
        this->blobs_[0]->gpu_data(),
        this->blobs_[0]->mutable_gpu_data());
    CUDA_POST_KERNEL_CHECK;
  } 

  Dtype step = this->blobs_[0]->mutable_cpu_data()[0];
  if (positive_) {
    min_ = Dtype(0);
    max_ = -(1 << (precision_ - 1)) * step;
  } else {
    min_ = -(1 << (precision_ - 1)) * step;
    max_ = ((1 << (precision_ - 1)) - 1) * step;
  }
  Quantize<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      bottom[0]->gpu_data(), count, step, min_, max_, 
      top[0]->mutable_gpu_data());
}

template <typename Dtype>
static __global__ void QuantizeClipGradients(const int n, Dtype min, Dtype max,
    const Dtype* in_data, const Dtype* in_diff, Dtype* out_diff) {
  CUDA_KERNEL_LOOP(i, n) {
    if (in_data[i] == min || in_data[i] == max) {
      out_diff[i] = Dtype(0);
    } else {
      out_diff[i] = in_diff[i];
    }
  }
}

template <typename Dtype>
void QuantizeLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[0]) {
    int count = top[0]->count();
    QuantizeClipGradients<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, min_, max_, 
        top[0]->gpu_data(), 
        top[0]->gpu_diff(), 
        bottom[0]->mutable_gpu_diff());
    CUDA_POST_KERNEL_CHECK;
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(QuantizeLayer);

}
